#include <iostream>
#include <vector>
#include <algorithm>

#include "rotate_array_kernel.cu"

using namespace std;

void cuda_rotate_array(vector<int>& nums, int k) {
    // If k is greater than the size of the array, we can just take the modulo
    k = k % nums.size();
    // Allocate memory on the GPU for the original array and result array
    int* d_nums;
    int* d_result;
    hipMalloc(&d_nums, nums.size() * sizeof(int));
    hipMalloc(&d_result, nums.size() * sizeof(int));

    // Copy the input array from host CPU to the GPU device
    hipMemcpy(d_nums, nums.data(), nums.size() * sizeof(int), hipMemcpyHostToDevice);
    
    // Define the number of threads per block and the number of blocks
    int threadsPerBlock = 256;
    int blocksPerGrid = (nums.size() + threadsPerBlock - 1) / threadsPerBlock;
    
    // Call the CUDA kernel to rotate the array
    rotateArrayKernel<<<blocksPerGrid, threadsPerBlock>>>(d_nums, d_result, nums.size(), k);

    // Copy the rotated array result from the GPU device back to the host CPU
    hipMemcpy(nums.data(), d_result, nums.size() * sizeof(int), hipMemcpyDeviceToHost);

    // Free the device memory
    hipFree(d_nums);
    hipFree(d_result);
}

void rotate_array_cuda_approach() {
    cout << "Rotate Array Approach 3: CUDA-based Rotation" << endl;
    vector<int> nums1 = {1, 2, 3, 4, 5, 6, 7};
    int k1 = 3;
    cuda_rotate_array(nums1, k1);
    for (int i = 0; i < nums1.size(); i++) {
        cout << nums1[i] << " ";
    }
    cout << endl;

    vector<int> nums2 = {-1, -100, 3, 99};
    int k2 = 2;
    cuda_rotate_array(nums2, k2);
    for (int i = 0; i < nums2.size(); i++) {
        cout << nums2[i] << " ";
    }
    cout << endl;
}

int main() {
    // CUDA-based Rotation Approach 3: using CUDA to rotate the array
    rotate_array_cuda_approach();

    return 0;
}
