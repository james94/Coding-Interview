#include <hip/hip_runtime.h>

__global__ void rotateArrayKernel(int* d_nums, int* d_result, int num_size, int k) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_size) {
        // Compute new index position to rotate to
        int new_rotate_pos = (idx + k) % num_size;
        // Store element into new position
        d_result[new_rotate_pos] = d_nums[idx];
    }
}
